#include<stdint.h>
#include<hip/hip_runtime.h>


#include <stdio.h>
#include <math.h>
#include <stdio.h>

#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)


//判断使用 单精度 or双精度 浮点数
#ifdef USE_DP
typedef double real;
#else
typedef float real;
#endif

const int NUM_REPEATS = 20;
// 计时
void timing(const real* x, const int N);
//数组元素求和 N为元素个数 x为数组地址
real reduce(const real* x, const int N);

int main(void)
{// 1亿个元素
    const int N = 100000000;
    //开辟空间，大小为 M bytes
    const int M = sizeof(real) * N;
    real* x = (real*)malloc(M);
    //初始化数组
    for (int n = 0; n < N; ++n)
    {
        x[n] = 1.23;
    }

    timing(x, N);

    free(x);
    return 0;
}

void timing(const real* x, const int N)
{
    real sum = 0;

    for (int repeat = 0; repeat < NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        sum = reduce(x, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    printf("sum = %f.\n", sum);
}

real reduce(const real* x, const int N)
{
    real sum = 0.0;
    for (int n = 0; n < N; ++n)
    {
        sum += x[n];
    }
    return sum;
}


